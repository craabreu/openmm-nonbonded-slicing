#include "hip/hip_runtime.h"
static __inline__ __device__ real2 multiplyComplex(real2 c1, real2 c2) {
    return make_real2(c1.x*c2.x-c1.y*c2.y, c1.x*c2.y+c1.y*c2.x);
}

/**
 * Load a value from the half-complex grid produces by a real-to-complex transform.
 */
static __inline__ __device__ real2 loadComplexValue(const real2* __restrict__ in, int x, int y, int z, int j) {
    const int inputZSize = ZSIZE/2+1;
    const int idist = XSIZE*YSIZE*inputZSize;
    if (z < inputZSize)
        return in[j*idist+x*YSIZE*inputZSize+y*inputZSize+z];
    int xp = (x == 0 ? 0 : XSIZE-x);
    int yp = (y == 0 ? 0 : YSIZE-y);
    real2 value = in[j*idist+xp*YSIZE*inputZSize+yp*inputZSize+(ZSIZE-z)];
    return make_real2(value.x, -value.y);
}

/**
 * Perform a 1D FFT on each row along one axis.
 */

extern "C" __global__ void execFFT(const INPUT_TYPE* __restrict__ in, OUTPUT_TYPE* __restrict__ out) {
    __shared__ real2 w[ZSIZE];
    __shared__ real2 data0[BLOCKS_PER_GROUP*ZSIZE];
    __shared__ real2 data1[BLOCKS_PER_GROUP*ZSIZE];
    for (int i = threadIdx.x; i < ZSIZE; i += blockDim.x)
        w[i] = make_real2(cos(-(SIGN)*i*2*M_PI/ZSIZE), sin(-(SIGN)*i*2*M_PI/ZSIZE));
    __syncthreads();

#if INPUT_IS_REAL
    const int idist = XSIZE*YSIZE*ZSIZE;
    const int odist = XSIZE*YSIZE*(ZSIZE/2+1);
#else
    const int idist = XSIZE*YSIZE*(ZSIZE/2+1);
    const int odist = XSIZE*YSIZE*ZSIZE;
#endif

    const int block = threadIdx.x/THREADS_PER_BLOCK;
    const int gridSize = XSIZE*YSIZE*ZSIZE;
    for (int baseIndex = blockIdx.x*BLOCKS_PER_GROUP; baseIndex < XSIZE*YSIZE; baseIndex += gridDim.x*BLOCKS_PER_GROUP) {
        int index = baseIndex+block;
        int x = index/YSIZE;
        int y = index-x*YSIZE;
        for (int j = 0; j < BATCH; j++) {
#if OUTPUT_IS_PACKED
            if (x < XSIZE/2+1) {
#endif
                if (index < XSIZE*YSIZE)
                    for (int i = threadIdx.x-block*THREADS_PER_BLOCK; i < ZSIZE; i += THREADS_PER_BLOCK)
#if INPUT_IS_REAL
                        data0[i+block*ZSIZE] = make_real2(in[j*idist+x*(YSIZE*ZSIZE)+y*ZSIZE+i], 0);
#elif INPUT_IS_PACKED
                        data0[i+block*ZSIZE] = loadComplexValue(in, x, y, i, j);
#else
                        data0[i+block*ZSIZE] = in[j*idist+x*(YSIZE*ZSIZE)+y*ZSIZE+i];
#endif
#if OUTPUT_IS_PACKED
            }
#endif
            __syncthreads();
            COMPUTE_FFT
        }
    }
}
